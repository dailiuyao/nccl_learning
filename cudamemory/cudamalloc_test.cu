#include <stdio.h>
#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cassert>


int main(int argc, char* argv[])
{
int datasize = 10;
float data_cpu[datasize];
float* sendbuff;
hipMalloc(&sendbuff, datasize * sizeof(float));

hipMemset(sendbuff, 0, datasize * sizeof(float));

hipMemcpy(data_cpu, sendbuff, data_size * sizeof(float), hipMemcpyDeviceToHost);

for(int i=0; i<datasize; i++)
    {        
        printf("data_cpu[%d] is: %f\n", i, data_cpu[i]);
    }

  return 0;
}